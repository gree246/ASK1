#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

hipError_t addWithCuda(unsigned long long int  liczba, bool &pierwsza);
bool CPU_sprawdz(unsigned long long int liczba);


#define NUM_THREADS   512
#define NUM_BLOCKS    1024

__global__ void addKernel(unsigned long long int *liczba, bool *pierwsza)
{
	if (*pierwsza == true) {
		unsigned long long int idx = 2 * (blockIdx.x*blockDim.x + threadIdx.x) + 3;
		while (idx*idx <= *liczba) {
			if (*liczba % (idx) == 0)  *pierwsza = false;
			if (*pierwsza == false) return;
			idx += blockDim.x*gridDim.x;
			if (idx*idx > *liczba) break;

		}

	}

}





int main()
{


	unsigned long long int liczba = 2 ^ 64 - 1;
	bool pierwsza;
	clock_t t1, t2;
	printf("Podaj liczbe:");
	scanf("%llu", &liczba);
	while (liczba > 18446744073709551615) {
		printf("Podaj liczbe:");
		scanf("%llu", &liczba);
	}
	t1 = clock();

	//pierwsza = CPU_sprawdz(liczba);

	t2 = clock();
	//printf("CPU   wynik: %d w %lf \n", pierwsza, (double)(t2-t1)/CLOCKS_PER_SEC);




	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(liczba, pierwsza);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}










	///printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//    c[0], c[1], c[2], c[3], c[4]);


	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.

	cudaStatus = hipDeviceReset();

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}


bool CPU_sprawdz(unsigned long long int liczba) {
	for (unsigned long long int i = 2; i < liczba; i++) {

		if (liczba % i == 0) {
			printf("%d , %d \n", liczba, i);
			return false;
		}

	}
	return true;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(unsigned long long int liczba, bool &pierwsza)
{
	pierwsza = true;
	unsigned long long int *dev_liczba = NULL;
	unsigned long long int *dev_zakres = NULL;
	bool *dev_pierwsza = NULL;
	hipError_t cudaStatus;
	clock_t t1, t2;
	float time_GPU = 0;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	if (liczba % 2 == 0) pierwsza = false;
	else {
		hipEvent_t start, stop;


		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0);

		// Allocate GPU buffers for three vectors (two input, one output)    .
		cudaStatus = hipMalloc((void**)&dev_liczba, sizeof(unsigned long long int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!1");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_pierwsza, sizeof(bool));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!2");
			goto Error;
		}




		// Copy input vectors from host memory to GPU buffers.
		cudaStatus = hipMemcpy(dev_liczba, &liczba, sizeof(unsigned long long int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed3!");
			goto Error;
		}


		cudaStatus = hipMemcpy(dev_pierwsza, &pierwsza, sizeof(bool), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!4");
			goto Error;
		}



		// Launch a kernel on the GPU with one thread for each element.






		addKernel << < NUM_BLOCKS, NUM_THREADS >> > (dev_liczba, dev_pierwsza);









		// Check for any errors launching the kernel
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}

		// Copy output vector from GPU buffer to host memory.
		cudaStatus = hipMemcpy(&pierwsza, dev_pierwsza, sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed5!");
			goto Error;
		}
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_GPU, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
	}




	printf("GPU wynik: %d w czasie: %LF ms \n", pierwsza, time_GPU);



Error:
	hipFree(dev_liczba);
	hipFree(dev_pierwsza);


	return cudaStatus;
}
